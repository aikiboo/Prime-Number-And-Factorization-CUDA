#include "hip/hip_runtime.h"
#include "gpu.hpp"



__global__ void isPrimeGPUV1(const ULONGLONG N,int*isPrime){
  int blockID = blockIdx.x+blockIdx.y*gridDim.x;
  ULONGLONG global_id = blockID* (blockDim.x*blockDim.y) + (threadIdx.y*blockDim.x) +threadIdx.x;
  while(global_id<N && isPrime[0]==1){
    if(isPrime[0]==1 && global_id>1 && N%global_id==0){
        isPrime[0] = 0;
        return;
    }
    global_id+=blockDim.x*blockDim.y*gridDim.x*gridDim.y;
  }
}

__global__ void isPrimeGPUV2(const ULONGLONG N,int*isPrime,const int* primes,const int sizeofPrimes){
  int blockID = blockIdx.x+blockIdx.y*gridDim.x;
  ULONGLONG global_id = blockID* (blockDim.x*blockDim.y) + (threadIdx.y*blockDim.x) +threadIdx.x;
  while(global_id<sizeofPrimes){
    if(N%primes[global_id]==0){
      isPrime[0] = 0;
    }
    global_id+=blockDim.x*blockDim.y*gridDim.x*gridDim.y;
  }
}



__host__ vector<ULONGLONG> searchPrimesGPUV1(const ULONGLONG N){
  vector<ULONGLONG> out = {2};
  for(ULONGLONG i=3;i<=N;i++){
    //init primes arrays
    int * primes;
    hipMalloc(&primes,sizeof(int)*out.size());
    hipMemcpy(primes,&out[0],sizeof(int),hipMemcpyHostToDevice);
    //init prime bool
    int* isPrimeArr;
    int isPrime[1] = {1};
    hipMalloc(&isPrimeArr,sizeof(int));
    hipMemcpy(isPrimeArr,isPrime,sizeof(int),hipMemcpyHostToDevice);
    //setup threads
    int threads = 32;
    dim3 nbthread(threads,threads);
    int x = ((out.size()+(threads-1))/threads+(threads-1))/threads;
    dim3 blocks(x,x);
    isPrimeGPUV2<<<blocks,nbthread>>>(i,isPrimeArr,primes,out.size());
    //receive if it's prime
    hipMemcpy( isPrime,isPrimeArr ,sizeof(bool), hipMemcpyDeviceToHost);
    if(isPrime[0]){
      //add to std::vector<int> v;
      out.push_back(i);
    }
  }
  return out;
}

__host__ bool isPrimeGPUlancherV1(const ULONGLONG N,ChronoGPU*chrGPU ){
  int* isPrimeArr;
  int isPrime[1] = {1};
  hipMalloc(&isPrimeArr,sizeof(int));
  hipMemcpy(isPrimeArr,isPrime,sizeof(int),hipMemcpyHostToDevice);
  int threads = 32;
  dim3 nbthread(threads,threads);
  int x = ((N+(threads-1))/threads+(threads-1))/threads;
  dim3 blocks(x,x);
  (*chrGPU).start();
  isPrimeGPUV1<<<blocks,nbthread>>>(N,isPrimeArr);
  (*chrGPU).stop();
  hipMemcpy(isPrime,isPrimeArr ,sizeof(bool), hipMemcpyDeviceToHost);
  return isPrime[0];
}
