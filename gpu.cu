#include "hip/hip_runtime.h"
#include "gpu.hpp"



__global__ void isPrimeGPUV1(const ULONGLONG N,int*isPrime){
  __shared__ int s_isPrime[1];
  ULONGLONG global_id = blockDim.x*blockIdx.x +threadIdx.x;
  if(threadIdx.x==0){
    s_isPrime[0]=1;
  }
  __syncthreads();
  while( (global_id%2==1 || global_id==2) && s_isPrime[0]==1 && global_id*global_id<=N){
    if(N%global_id==0 && global_id>1 ){
        s_isPrime[0] = 0;
        break;
    }
    global_id+=blockDim.x*gridDim.x;
  }
  __syncthreads();
  if(threadIdx.x==0 && s_isPrime[0]==0){
    isPrime[0]=s_isPrime[0];
  }
}

__host__ bool isPrimeGPUlancherV1(const ULONGLONG N,ChronoGPU*chrGPU ){
  int* isPrimeArr;
  int isPrime[1] = {1};
  hipMalloc(&isPrimeArr,sizeof(int));
  hipMemcpy(isPrimeArr,isPrime,sizeof(int),hipMemcpyHostToDevice);
  int threads = NB_THREADS;
  int blocks = (sqrt(N)+NB_THREADS-1)/NB_THREADS;
  (*chrGPU).start();
  isPrimeGPUV1<<<1,threads>>>(N,isPrimeArr);
  (*chrGPU).stop();
  hipMemcpy(isPrime,isPrimeArr ,sizeof(bool), hipMemcpyDeviceToHost);
  hipFree(isPrimeArr);
  return isPrime[0];
}

__global__ void searchPrimesGPUV1(const ULONGLONG N,char* primes){
  ULONGLONG global_id = blockIdx.x*blockDim.x +threadIdx.x;
  ULONGLONG val = (global_id*2)+3;
  if(global_id>N)
    return;
  if(primes[global_id]==0){
    for(ULONGLONG x=global_id+val;x<N;x+=val){
      primes[x]=1;
    }
  }

}

__host__ vector<ULONGLONG> searchPrimesGPUV1Launcher(const ULONGLONG N,ChronoGPU*chrGPU){
  vector<ULONGLONG> out = {2};
  int nbArrEl = sqrt(N)/2 + 1;
  int sizeArr = sizeof(char)*nbArrEl;
  char* isPrimeArr_dev;
  char* isPrimeArr = (char*) malloc(sizeArr);
  hipMalloc(&isPrimeArr_dev,sizeArr);
  int threads = NB_THREADS;
  int blocks = (nbArrEl+NB_THREADS-1)/NB_THREADS;
  (*chrGPU).start();
  searchPrimesGPUV1<<<blocks,threads>>>(nbArrEl,isPrimeArr_dev);
  (*chrGPU).stop();
  hipMemcpy(isPrimeArr,isPrimeArr_dev ,sizeArr, hipMemcpyDeviceToHost);
  hipFree(isPrimeArr_dev);
  for(ULONGLONG x=0;x<nbArrEl;x++){
    if(!isPrimeArr[x]){
      out.push_back(x*2+3);
    }
  }
  return out;
}


__global__ void FactorizationGPUV1(const ULONGLONG N,const ULONGLONG* primes,const ULONGLONG primesSize,char* coefs){
    ULONGLONG global_id = blockIdx.x*blockDim.x +threadIdx.x;
    __shared__ char s_coefs[NB_THREADS];
    s_coefs[global_id]=0;
    if(global_id>primesSize)
      return;
    ULONGLONG val = primes[global_id];
    if(N%val){
        char coef =0;
        ULONGLONG tmp = N;
        while(tmp%val==0){
          coef++;
          tmp/=val;
        }
        s_coefs[global_id]=coef;
    }
    coefs[global_id]=s_coefs[global_id];

}

__host__ void FactorizationGPUV1Launcher(const ULONGLONG N,ChronoGPU*chrGPU,vector<ULONGLONG>* primes,vector<Cell> *cells){
  ULONGLONG* primeArr_dev;
  int nbArrEl = primes->size();
  int sizeArr = sizeof(char)*nbArrEl;
  char* coefs_devs;
  char* coefs = (char*)malloc(sizeArr);
  hipMalloc(&coefs_devs,sizeArr);
  hipMalloc(&primeArr_dev,sizeof(ULONGLONG)*nbArrEl);
  hipMemcpy(primeArr_dev,&primes[0] ,sizeof(ULONGLONG)*nbArrEl, hipMemcpyHostToDevice);
  int threads = NB_THREADS;
  int blocks = (nbArrEl+NB_THREADS-1)/NB_THREADS;
  (*chrGPU).start();
  FactorizationGPUV1<<<blocks,threads>>>(N,primeArr_dev,nbArrEl,coefs_devs);
  (*chrGPU).stop();
  hipMemcpy(coefs,coefs_devs ,sizeof(char)*nbArrEl, hipMemcpyDeviceToHost);
  for(int i =0;i<nbArrEl;i++){
    if(coefs[i]>0){
      Cell cell;
      cell.expo = coefs[i];
      cell.value = primes->at(i);
      cells->push_back(cell);
    }
  }
  if(cells->size()==0){
    Cell cell;
    cell.expo = 1;
    cell.value = N;
    cells->push_back(cell);
  }
}
